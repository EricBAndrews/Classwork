/*
  Starting file for the cuda version.
  You do not have to do the same multple level block for this to work

*/


#include <hip/hip_runtime.h>
#include<stdio.h>

const char* dgemm_desc = "CUDA dgemm";

#if !defined(BLOCK_SIZE)
#define BLOCK_SIZE 10
#endif

#if !defined(CUDA_BLOCK_SIZE)
#define CUDA_BLOCK_SIZE 16
#endif

#define min(a,b) (((a)<(b))?(a):(b))

// Perform smaller dgemm operation using CUDA
__global__ void do_block(int lda, int i, int j, int k, double*matA, double* matB,
			 double* matC, int M, int N) {
  /* int row = i + threadIdx.x + (blockIdx.x * CUDA_BLOCK_SIZE); */
  /* int col = j + threadIdx.y + (blockIdx.y * CUDA_BLOCK_SIZE); */
  // row, col within block
  int locRow = threadIdx.x / BLOCK_SIZE;
  int locCol = threadIdx.x % BLOCK_SIZE;
  // row, col within full matrix
  int glbRow = i + locRow;
  int glbCol = j + locCol;
  //int lBound = min(BLOCK_SIZE, lda-glbRow);
  //if(threadIdx.x == 0) printf("lBound: %i\n", lBound);
  double a;
  double b;
  double cij;
  //if(threadIdx.x < BLOCK_SIZE * BLOCK_SIZE) {
  //if(glbRow < lda && glbCol < lda) {
  for (int l = 0; l < BLOCK_SIZE; ++l) {
    //if (locRow < M && locCol < N) {
    //printf("i: %i | j: %i | locRow: %i | locCol: %i | glbRow: %i | glbCol: %i | M: %i | N: %i\n", i, j, locRow, locCol, glbRow, glbCol, M, N);
    cij = 0;
    //for (int l = 0; l < lBound; ++l) {
    //printf("matB[%i][%i]: %f\n", l, glbCol, matB[l*lda + glbCol]);
    if ((i+l) < M && (j+l) < N) {
      if (glbRow == 0 && glbCol == 0) printf("M: %i | N: %i | glbRow+l: %i | glbCol+l: %i\n", M, N, glbRow+l, glbCol+l);
      a = matA[glbRow*lda + i+l];
      b = matB[(j+l)*lda + glbCol];
      cij += a * b;
      if (glbRow == 0 && glbCol == 0) printf("i: %i | j: %i | l: %i | cij: %f | matC[0][0]: %f\n", i, j, l, cij, matC[0]);
    }
    matC[glbRow*lda + glbCol] += cij;
    //if (glbRow == 0 && glbCol ==0 ) printf("matC: %.1f\n", matC[glbRow*lda + glbCol]);
  }
}

__global__ void printDeviceMatrix(double* mat, int lda) {
  for (int i = 0; i < lda; ++i) {
    for (int j = 0; j < lda; ++j) {
      if (mat[i*lda + j] > 0) printf(" ");
      printf("%.1f ", mat[i*lda + j]);
    }
    printf("\n");
  }
}
/* This routine performs a dgemm operation
 *  C := C + A * B
 * where A, B, and C are lda-by-lda matrices stored in column-major format. 
 * On exit, A and B maintain their input values. */  
void square_dgemm (int lda, double* A, double* B, double* C) {
  // transfer the matrices over, allocate size for C
  size_t matSize = lda * lda * sizeof(double);
  double* matA;
  double* matB;
  double* matC;
  hipMalloc(&matA, matSize);
  hipMalloc(&matB, matSize);
  hipMalloc(&matC, matSize);
  hipMemcpy(matA, A, matSize, hipMemcpyHostToDevice);
  hipMemcpy(matB, B, matSize, hipMemcpyHostToDevice);
  /* for (int i = 0; i < lda; ++i) { */
  /*   for (int j = 0; j < lda; ++j) { */
  /*     if (A[i*lda + j] > 0) printf(" "); */
  /*     printf("%.1f ", A[i*lda + j]); */
  /*   } */
  /*   printf("\n"); */
  /* } */
  //printDeviceMatrix<<<1, 1>>>(matA, lda);
  //printDeviceMatrix<<<1, 1>>>(matB, lda);
  //printDeviceMatrix<<<1, 1>>>(matC, lda);
  /* For each block-row of A */ 
  for (int i = 0; i < lda; i += BLOCK_SIZE) {
    /* For each block-column of B */
    for (int j = 0; j < lda; j += BLOCK_SIZE) {
      /* Accumulate block dgemms into block of C */
      for (int k = 0; k < lda; k += BLOCK_SIZE) {
	/* Correct block dimensions if block "goes off edge of" the matrix */
	int M = min(BLOCK_SIZE, lda-i);
	int N = min(BLOCK_SIZE, lda-j);
	// dimGrid = threads per block
	// dimBlock = block layout
	
	/* dim3 dimBlock(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE); */

	/* int gridSize = BLOCK_SIZE / CUDA_BLOCK_SIZE; // 2 */
	/* if (gridSize * CUDA_BLOCK_SIZE < BLOCK_SIZE) { // 32 < 41 */
	/*   ++gridSize; */
	/* } */
	/* dim3 dimGrid(gridSize, gridSize); */
	

	/* Perform individual block dgemm */
	// printf("i: %i; j: %i; k: %i\n", i, j, k);
	do_block<<<1, 256>>>(lda, i, j, k, matA, matB, matC, M, N);
	// do_block(lda, M, N, K, A + i + k*lda, B + k + j*lda, C + i + j*lda);
      }
    }
  }
  hipDeviceSynchronize();
  hipMemcpy(C, matC, matSize, hipMemcpyDeviceToHost);
  //printDeviceMatrix<<<1, 1>>>(matC, lda);
  /* for (int i = 0; i < lda; ++i) { */
  /*   for (int j = 0; j < lda; ++j) { */
  /*     printf("%.1f ", C[i*lda + j]); */
  /*   } */
  /*   printf("\n"); */
  /* } */
}
