/*
  Starting file for the cuda version.
  You do not have to do the same multple level block for this to work

*/


#include <hip/hip_runtime.h>
#include<stdio.h>

const char* dgemm_desc = "CUDA dgemm";

#if !defined(BLOCK_SIZE)
#define BLOCK_SIZE 512
#endif

#if !defined(CUDA_BLOCK_SIZE)
#define CUDA_BLOCK_SIZE 16
#endif

#if !defined(CUDA_GRID_SIZE)
#define CUDA_GRID_SIZE 32
#endif

#define min(a,b) (((a)<(b))?(a):(b))

// Perform smaller dgemm operation using CUDA
__global__ void do_block(int lda, double*matA, double* matB, double* matC, int M, int N, int K) {
  int row = threadIdx.y + blockIdx.y*CUDA_BLOCK_SIZE;
  int col = threadIdx.x + blockIdx.x*CUDA_BLOCK_SIZE;;
  double c;
  //double a, b, c;
  //printf("blockIdx.x: %i | blockIdx.y: %i | gridDim.x: %i | gridDim.y: %i\n", blockIdx.x, blockIdx.y, gridDim.x, gridDim.y);
  if (row < M && col < N) {
    //printf("row: %i | col: %i\n", row, col);
    c = matC[row + col*lda];
    for (int k = 0; k < K; ++k) {
      //a = matA[k*lda + row];
      //b = matB[col*lda + k];
      //c += a * b;
      c += matA[k*lda + row] * matB[col*lda + k];
    }
    matC[row + col*lda] = c;
  }
}

__global__ void printDeviceMatrix(double* mat, int lda) {
  for (int i = 0; i < lda; ++i) {
    for (int j = 0; j < lda; ++j) {
      if (mat[i*lda + j] > 0) printf(" ");
      printf("%.0f ", mat[i*lda + j]);
    }
    printf("\n");
  }
}
/* This routine performs a dgemm operation
 *  C := C + A * B
 * where A, B, and C are lda-by-lda matrices stored in column-major format. 
 * On exit, A and B maintain their input values. */  
void square_dgemm (int lda, double* A, double* B, double* C) {
  // transfer the matrices over, allocate size for C
  size_t matSize = lda * lda * sizeof(double);
  double* matA;
  double* matB;
  double* matC;
  hipMalloc(&matA, matSize);
  hipMalloc(&matB, matSize);
  hipMalloc(&matC, matSize);
  hipMemcpy(matA, A, matSize, hipMemcpyHostToDevice);
  hipMemcpy(matB, B, matSize, hipMemcpyHostToDevice);
  /* For each block-row of A */ 
  for (int i = 0; i < lda; i += BLOCK_SIZE) {
    /* For each block-column of B */
    for (int j = 0; j < lda; j += BLOCK_SIZE) {
      /* Accumulate block dgemms into block of C */
      for (int k = 0; k < lda; k += BLOCK_SIZE) {
	/* Correct block dimensions if block "goes off edge of" the matrix */
	int M = min(BLOCK_SIZE, lda-i);
	int N = min(BLOCK_SIZE, lda-j);
	int K = min(BLOCK_SIZE, lda-k);

	//int gridSize = BLOCK_SIZE / CUDA_BLOCK_SIZE;

	//dim3 gridDim(gridSize, gridSize);
	dim3 gridDim(CUDA_GRID_SIZE, CUDA_GRID_SIZE);
	dim3 blockDim(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);

	/* Perform individual block dgemm */
	// printf("i: %i; j: %i; k: %i\n", i, j, k);
	do_block<<<gridDim, blockDim>>>(lda, matA + k*lda + i, matB + j*lda + k, matC + i + j*lda, M, N, K);
      }
    }
  }
  hipDeviceSynchronize();
  hipMemcpy(C, matC, matSize, hipMemcpyDeviceToHost);
  //printDeviceMatrix<<<1, 1>>>(matC, lda);
  /* for (int i = 0; i < lda; ++i) { */
  /*   for (int j = 0; j < lda; ++j) { */
  /*     printf("%.2f ", C[i*lda + j]); */
  /*   } */
  /*   printf("\n"); */
  /* } */
}
