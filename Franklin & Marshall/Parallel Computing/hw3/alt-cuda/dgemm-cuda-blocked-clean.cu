
#include <hip/hip_runtime.h>
/*
Starting file for the cuda version.
You do not have to do the same multple level block for this to work

*/

const char* dgemm_desc = "CUDA dgemm";


/* This routine performs a dgemm operation
 *  C := C + A * B
 * where A, B, and C are lda-by-lda matrices stored in column-major format. 
 * On exit, A and B maintain their input values. */  
void square_dgemm (int lda, double* A, double * B, double* C)
{
	//fill me
}
