/*
  Starting file for the cuda version.
  You do not have to do the same multple level block for this to work

*/


#include <hip/hip_runtime.h>
#include<stdio.h>

const char* dgemm_desc = "CUDA dgemm";

#if !defined(BLOCK_SIZE)
#define BLOCK_SIZE 32
#endif

#if !defined(CUDA_BLOCK_SIZE)
#define CUDA_BLOCK_SIZE 16
#endif

#define min(a,b) (((a)<(b))?(a):(b))

// Perform smaller dgemm operation using CUDA
__global__ void do_block(int lda, double*matA, double* matB, double* matC, int M, int N, int K) {
  int row = threadIdx.x % M;
  int col = threadIdx.x / M;
  double a, b, c;
  if (row < M && col < N) {
    c = matC[row*lda + col];
    for (int k = 0; k < K; ++k) {
      a = matA[k*lda + row];
      b = matB[col*lda + k];
      c += a * b;
    }
    matC[row*lda + col] = c;
  }
}

__global__ void printDeviceMatrix(double* mat, int lda) {
  for (int i = 0; i < lda; ++i) {
    for (int j = 0; j < lda; ++j) {
      if (mat[i*lda + j] > 0) printf(" ");
      printf("%.0f ", mat[i*lda + j]);
    }
    printf("\n");
  }
}
/* This routine performs a dgemm operation
 *  C := C + A * B
 * where A, B, and C are lda-by-lda matrices stored in column-major format. 
 * On exit, A and B maintain their input values. */  
void square_dgemm (int lda, double* A, double* B, double* C) {
  // transfer the matrices over, allocate size for C
  size_t matSize = lda * lda * sizeof(double);
  double* matA;
  double* matB;
  double* matC;
  hipMalloc(&matA, matSize);
  hipMalloc(&matB, matSize);
  hipMalloc(&matC, matSize);
  hipMemcpy(matA, A, matSize, hipMemcpyHostToDevice);
  hipMemcpy(matB, B, matSize, hipMemcpyHostToDevice);
  /* For each block-row of A */ 
  for (int i = 0; i < lda; i += BLOCK_SIZE) {
    /* For each block-column of B */
    for (int j = 0; j < lda; j += BLOCK_SIZE) {
      /* Accumulate block dgemms into block of C */
      for (int k = 0; k < lda; k += BLOCK_SIZE) {
	/* Correct block dimensions if block "goes off edge of" the matrix */
	int M = min(BLOCK_SIZE, lda-i);
	int N = min(BLOCK_SIZE, lda-j);
	int K = min(BLOCK_SIZE, lda-k);

	/* Perform individual block dgemm */
	// printf("i: %i; j: %i; k: %i\n", i, j, k);
	do_block<<<1, 1024>>>(lda, matA + k*lda + i, matB + j*lda +k, matC + i*lda + j, M, N, K);
      }
    }
  }
  hipDeviceSynchronize();
  hipMemcpy(C, matC, matSize, hipMemcpyDeviceToHost);
  //printDeviceMatrix<<<1, 1>>>(matC, lda);
  /* for (int i = 0; i < lda; ++i) { */
  /*   for (int j = 0; j < lda; ++j) { */
  /*     printf("%.2f ", C[i*lda + j]); */
  /*   } */
  /*   printf("\n"); */
  /* } */
}
